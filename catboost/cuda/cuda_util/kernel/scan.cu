#include "scan.cuh"
#include "segmented_scan_helpers.cuh"
#include "fill.cuh"

#include <contrib/libs/nvidia/cub/cub/device/device_scan.cuh>
#include <contrib/libs/nvidia/cub/cub/iterator/transform_input_iterator.cuh>

namespace NKernel {

    template <typename T, typename TOut>
    hipError_t ScanVector(const T* input, TOut* output, ui32 size, bool inclusive, TScanKernelContext<T, TOut>& context, TCudaStream stream) {
        using TKernelContext = TScanKernelContext<T, TOut>;

        if (inclusive) {
            return hipcub::DeviceScan::InclusiveSum(context.PartResults, context.NumParts, input, output, size, stream);
        } else {
            return hipcub::DeviceScan::ExclusiveSum(context.PartResults, context.NumParts, input, output, size, stream);
        }
    }

    template <class T>
    struct TToSignedConversion {
        using TSignedType = T;
    };


    template <>
    struct TToSignedConversion<ui32> {
        using TSignedType = int;
    };

    template <typename T_, typename TOut_>
    hipError_t SegmentedScanNonNegativeVector(const T_* input, TOut_* output, ui32 size, bool inclusive, TScanKernelContext<T_, TOut_>& context, TCudaStream stream) {
        using TKernelContext = TScanKernelContext<T_, TOut_>;
        using T = typename TToSignedConversion<T_>::TSignedType;
        using TOut = typename TToSignedConversion<TOut_>::TSignedType;
        T zeroValue = 0.0f;
        if (inclusive) {
            return hipcub::DeviceScan::InclusiveScan((TOut*)context.PartResults.Get(), context.NumParts, (const T*)input, (TOut*)output, TNonNegativeSegmentedSum(), size, stream);
        } else {
            return hipcub::DeviceScan::ExclusiveScan((TOut*)context.PartResults.Get(), context.NumParts, (const T*) input, (TOut*)output, TNonNegativeSegmentedSum(), zeroValue, size, stream);
        }
    }


    template <typename T_>
    hipError_t SegmentedScanAndScatterNonNegativeVector(const T_* input, const ui32* indices, T_* output,
                                                         ui32 size, bool inclusive,
                                                         TScanKernelContext<T_, T_>& context,
                                                         TCudaStream stream) {
        using TKernelContext = TScanKernelContext<T_, T_>;
        using T = typename TToSignedConversion<T_>::TSignedType;

        if (inclusive) {
            TNonNegativeSegmentedScanOutputIterator<hipcub::STORE_CS, T,  ptrdiff_t, true>  outputIterator((T*)output, indices, indices + size);
            return hipcub::DeviceScan::InclusiveScan((T*)context.PartResults.Get(), context.NumParts, (const T*)input, outputIterator, TNonNegativeSegmentedSum(), size, stream);
        } else {
            TNonNegativeSegmentedScanOutputIterator<hipcub::STORE_CS, T,  ptrdiff_t, false>  outputIterator((T*)output, indices, indices + size);
            FillBuffer<T>((T*)output, 0, size, stream);
            return hipcub::DeviceScan::InclusiveScan((T*)context.PartResults.Get(), context.NumParts, (const T*) input, outputIterator, TNonNegativeSegmentedSum(), size, stream);
        }
    }

    template <class T, class TOut>
    ui64 ScanVectorTempSize(ui32 size, bool inclusive) {
        ui64 sizeInBytes = 0;
        if (inclusive) {
            hipcub::DeviceScan::InclusiveSum<const T*, TOut*>(nullptr, sizeInBytes, nullptr, nullptr, size);
        } else {
            hipcub::DeviceScan::ExclusiveSum<const T*, TOut*>(nullptr, sizeInBytes, nullptr, nullptr, size);
        }
        return sizeInBytes;
    }



    #define SCAN_VECTOR(Type, TypeOut) \
    template  hipError_t ScanVector<Type, TypeOut>(const Type *input, TypeOut *output, ui32 size, bool inclusive, TScanKernelContext<Type, TypeOut>& context, TCudaStream stream); \
    template  hipError_t SegmentedScanNonNegativeVector<Type>(const Type *input, TypeOut *output, ui32 size, bool inclusive, TScanKernelContext<Type, TypeOut>& context, TCudaStream stream); \
    template ui64 ScanVectorTempSize<Type, TypeOut>(ui32, bool);

    SCAN_VECTOR(int, int)
    SCAN_VECTOR(ui32, ui32)
    SCAN_VECTOR(float, float)
    SCAN_VECTOR(double, double)

    namespace {
        struct TCastToUi64 {
            template <typename InputT>
            __host__ __device__
            ui64 operator()(InputT v) const
            {
                return static_cast<ui64>(v);
            }
        };
        using TUi32AsUi64 = hipcub::TransformInputIterator<ui64, TCastToUi64, ui32*>;
    }

    template <>
    hipError_t ScanVector<ui32, ui64>(const ui32* input, ui64* output, ui32 size, bool inclusive, TScanKernelContext<ui32, ui64>& context, TCudaStream stream) {
        TUi32AsUi64 inputAsUi64(const_cast<ui32*>(input), TCastToUi64());
        if (inclusive) {
            return hipcub::DeviceScan::InclusiveSum(context.PartResults, context.NumParts, inputAsUi64, output, size, stream);
        } else {
            return hipcub::DeviceScan::ExclusiveSum(context.PartResults, context.NumParts, inputAsUi64, output, size, stream);
        }
    }

    template <>
    ui64 ScanVectorTempSize<ui32, ui64>(ui32 size, bool inclusive) {
        ui64 sizeInBytes = 0;
        if (inclusive) {
            hipcub::DeviceScan::InclusiveSum<TUi32AsUi64, ui64*>(nullptr, sizeInBytes, TUi32AsUi64(nullptr, TCastToUi64()), nullptr, size);
        } else {
            hipcub::DeviceScan::ExclusiveSum<TUi32AsUi64, ui64*>(nullptr, sizeInBytes, TUi32AsUi64(nullptr, TCastToUi64()), nullptr, size);
        }
        return sizeInBytes;
    }

    template <>
    hipError_t SegmentedScanNonNegativeVector<ui32, ui64>(const ui32* input, ui64* output, ui32 size, bool inclusive, TScanKernelContext<ui32, ui64>& context, TCudaStream stream) {
        CB_ENSURE_INTERNAL(false, "This function should never be called");
        return hipErrorUnknown;
    }

    #define SEGMENTED_SCAN_VECTOR(Type) \
    template  hipError_t SegmentedScanAndScatterNonNegativeVector<Type>(const Type *input, const ui32* indices, Type *output, ui32 size, bool inclusive, TScanKernelContext<Type, Type>& context, TCudaStream stream);

    SEGMENTED_SCAN_VECTOR(int)
    SEGMENTED_SCAN_VECTOR(ui32)
    SEGMENTED_SCAN_VECTOR(float)
    SEGMENTED_SCAN_VECTOR(double)


}
