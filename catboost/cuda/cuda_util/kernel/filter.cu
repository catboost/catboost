#include "hip/hip_runtime.h"
#include "filter.cuh"

namespace NKernel {

    struct TZeroWeightFilter {

        __device__ ui32 operator()(float w) {
            return abs(w) > 1e-20f;
        }
    };

    template <class Filter = TZeroWeightFilter>
    __global__ void FilterImpl(const float* weights,
                               ui32 size,
                               ui32* result) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        Filter filter;
        if (i < size) {
            result[i] = filter(__ldg(weights + i));
        }
    }


    void Filter(const float* weights, const ui32 size, ui32* result, TCudaStream stream) {
        if (size > 0) {
            const ui32 blockSize = 512;
            const ui32 numBlocks = (size + blockSize - 1) / (blockSize);
            FilterImpl << <numBlocks, blockSize, 0, stream>>>(weights, size, result);
        }
    }
}
