#include "hip/hip_runtime.h"
#include "compression.cuh"
#include "compression_helper.cuh"
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <library/cpp/cuda/wrappers/arch.cuh>
#include <stdio.h>
#include <stdlib.h>



//TODO: if we'll need more memory, try full block bit compression instead of ui64 as storage for keys
namespace NKernel {

    template <class TStorageType, ui32 BLOCK_SIZE>
    __global__ void CompressImpl(const ui32* src, int size, TStorageType* dst, ui32 bitsPerKey, int blockCount) {

        int blockId = blockIdx.x;
        TCompressionHelper<TStorageType, BLOCK_SIZE> helper(bitsPerKey);
        while (blockId < blockCount) {
            TDirectLoader loader(src + helper.KeysPerBlock() * blockId);
            helper.CompressBlock(loader, size - helper.KeysPerBlock() * blockId, dst + BLOCK_SIZE * blockId);
            blockId += gridDim.x;
        }
    }


    template <class TStorageType, ui32 BLOCK_SIZE>
    __global__ void DecompressImpl(const TStorageType* src, ui32* dst, int size, ui32 bitsPerKey, int blockCount) {

        TCompressionHelper<TStorageType, BLOCK_SIZE> helper(bitsPerKey);

        int blockId = blockIdx.x;
        while (blockId < blockCount) {
            TDirectWriter writer(dst + helper.KeysPerBlock() * blockId);
            helper.DecompressBlock(writer, src +  BLOCK_SIZE * blockId, size - helper.KeysPerBlock() * blockId);
            blockId += gridDim.x;
        }
    }


    template <class TStorageType, ui32 BLOCK_SIZE>
    __global__ void GatherFromCompressedImpl(const TStorageType* src,
                                             const ui32* map, ui32 mapMask,
                                             ui32* dst, int size, ui32 bitsPerKey) {

        TCompressionHelper<TStorageType, BLOCK_SIZE> helper(bitsPerKey);
        ui32 tid = blockIdx.x * blockDim.x + threadIdx.x;
        while (tid < size) {
            const ui32 value = helper.Read(src, map[tid] & mapMask);
            dst[tid] = value;
            tid += blockDim.x * gridDim.x;
        }
    }

    template <class TStorageType>
    void GatherFromCompressed(const TStorageType* src, const ui32* map, ui32 mapMask, ui32* dst, ui32 size, ui32 bitsPerKey, TCudaStream stream) {

        constexpr ui32 compressedBlockSize = CompressCudaBlockSize();
        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);

        GatherFromCompressedImpl<TStorageType, compressedBlockSize> << < min(TArchProps::MaxBlockCount(), numBlocks), blockSize, 0, stream >> >(src, map, mapMask, dst, size, bitsPerKey);
    }


    template <class TStorageType>
    void Decompress(const TStorageType* src, ui32* dst, ui32 size, ui32 bitsPerKey, TCudaStream stream) {

        constexpr ui32 blockSize = CompressCudaBlockSize();
        const ui32 numBlocks = CeilDivide((int)size, TCompressionHelper<TStorageType, blockSize>(bitsPerKey).KeysPerBlock());

        DecompressImpl<TStorageType, blockSize> << < min(TArchProps::MaxBlockCount(), numBlocks), blockSize, 0, stream >> >(src, dst, size, bitsPerKey, numBlocks);
    }

    template <class TStorageType>
    void Compress(const ui32* src, TStorageType* dst,  ui32 size, ui32 bitsPerKey, TCudaStream stream) {

        constexpr ui32 blockSize = CompressCudaBlockSize();
        const ui32 numBlocks = CeilDivide((int)size, TCompressionHelper<TStorageType, blockSize>(bitsPerKey).KeysPerBlock());
        CompressImpl<TStorageType, blockSize> << < min(TArchProps::MaxBlockCount(), numBlocks), blockSize, 0, stream >> >(src, size, dst, bitsPerKey, numBlocks);
    }

    #define COMPRESS(Type) \
    template void GatherFromCompressed<Type>(const Type* src, const ui32* map, ui32 mapMask, ui32* dst, ui32 size, ui32 bitsPerKey, TCudaStream stream); \
    template void Compress<Type>(const ui32* src, Type* dst, ui32 size, ui32 bitsPerKey, TCudaStream stream);\
    template void Decompress<Type>(const Type* src, ui32* dst,  ui32 size, ui32 bitsPerKey, TCudaStream stream);

    COMPRESS(ui32)
    COMPRESS(ui64)


}




