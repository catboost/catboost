#include "hip/hip_runtime.h"
#include "update_part_props.cuh"
#include "fill.cuh"
#include <library/cpp/cuda/wrappers/arch.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <catboost/cuda/cuda_util/gpu_data/partitions.h>


namespace NKernel {



    template <int BlockSize, int N = 1>
    __forceinline__ __device__  double ComputeSum(const float* __restrict__ stat, ui32 offset, ui32 size, int blockIdx, int blockCount) {

        float4 sum;
        sum.x =  sum.y = sum.z = sum.w = 0;

        stat += offset;

        const int warpSize = 32;
        const int alignSize = 4 * warpSize;

        {
            int lastId = min(size, alignSize - (offset % alignSize));

            if (blockIdx == 0) {
                if (threadIdx.x < lastId) {
                    sum.x += Ldg(stat + threadIdx.x);
                }
            }

            size = max(size - lastId, 0);
            stat += lastId;
        }

        //now lets align end
        const int unalignedTail = (size % alignSize);

        if (unalignedTail != 0) {
            if (blockIdx == 0) {
                const int tailOffset = size - unalignedTail;
                if (threadIdx.x < unalignedTail) {
                    sum.y += Ldg(stat + tailOffset + threadIdx.x);
                }
            }
        }
        size -= unalignedTail;

        const int entriesPerWarp = warpSize * 4;
        const int warpsPerBlock = (BlockSize / 32);
        const int globalWarpId = (blockIdx * warpsPerBlock) + (threadIdx.x / 32);
        stat += globalWarpId * entriesPerWarp;
        size = max(size - globalWarpId * entriesPerWarp, 0);

        const int stripeSize = entriesPerWarp * warpsPerBlock * blockCount;

        const int localIdx = (threadIdx.x & 31) * 4;
        const int iterCount = (size - localIdx + stripeSize - 1)  / stripeSize;

        stat += localIdx;
        double accumResult = 0;
        const int M = 8;
        if (size > 0) {
            int i = 0;
            for (; i <= iterCount - N * M; i += N * M) {
                #pragma unroll 4
                for (int j = 0; j < N * M; ++j) {
                    const float4* stat4 = (const float4*) stat;
                    float4 val = Ldg(stat4);
                    sum.x += val.x;
                    sum.y += val.y;
                    sum.z += val.z;
                    sum.w += val.w;
                    stat += stripeSize;
                }
                accumResult += (double)sum.x + (double)sum.y + (double)sum.z + (double)sum.w;
                sum = {0};
            }
            #pragma unroll N
            for (; i < iterCount; ++i) {
                const float4* stat4 = (const float4*) stat;
                float4 val = Ldg(stat4);
                sum.x += val.x;
                sum.y += val.y;
                sum.z += val.z;
                sum.w += val.w;
                stat += stripeSize;
            }
        }

        return accumResult + (double)sum.x + (double)sum.y + (double)sum.z + (double)sum.w;
    };


    template <class TOutput>
    __global__ void SaveResultsImpl(const ui32* partIds,
                                    const double* tempVars,
                                    ui32 partCount,
                                    ui32 statCount,
                                    int tempVarsBlockCount,
                                    TOutput* statSums) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        const ui32 statId = i % statCount;
        const ui32 y =  i / statCount;
        if (i < partCount * statCount) {
            const ui32 leafId = partIds != nullptr ? partIds[y] : y;
            double total = 0;
            for (int x = 0; x < tempVarsBlockCount; ++x) {
                total += tempVars[i];
                tempVars += statCount * partCount;
            }
            statSums[leafId * statCount + statId] = total;
        }
    }


    template <int BlockSize>
    __launch_bounds__(BlockSize, 2)
    __global__ void UpdatePartitionsPropsForOffsetsImpl(const ui32* offsets,
                                                        const float* source,
                                                        ui64 statLineSize,
                                                        ui32 partCount,
                                                        double* statPartSums) {

        ui32 partId = blockIdx.y;
        const ui32 statId = blockIdx.z;
        source += statId * statLineSize;

        while (partId < partCount) {

            const ui32 partOffset = __ldg(offsets + partId);
            const ui32 partSize = __ldg(offsets + partId + 1) - partOffset;

            __shared__ volatile double localBuffer[BlockSize];

            const int minDocsPerBlock = BlockSize * 16;
            const int effectiveBlockCount = min(gridDim.x, (partSize + minDocsPerBlock - 1) / minDocsPerBlock);

            double result = 0;

            if (blockIdx.x < effectiveBlockCount) {
                const int blockId = blockIdx.x % effectiveBlockCount;

                localBuffer[threadIdx.x] = ComputeSum < BlockSize > (source, partOffset, partSize, blockId, effectiveBlockCount);
                __syncthreads();

                result = FastInBlockReduce(threadIdx.x, localBuffer, BlockSize);
            }

            if (threadIdx.x == 0) {
                const int statCount = gridDim.z;
                const int lineSize = statCount * partCount;
                ui64 idx = blockIdx.x * lineSize + partId * statCount + statId;
                statPartSums[idx] = result;
            }

            partId += gridDim.y;
        }
    }

    template <int BlockSize>
    __launch_bounds__(BlockSize, 2)
    __global__ void UpdatePartitionsPropsImpl(const ui32* partIds,
                                              const TDataPartition* parts,
                                              const float* source,
                                              ui64 statLineSize,
                                              double* tempVars) {
        const ui32 leafId = partIds[blockIdx.y];
        TDataPartition part = parts[leafId];

        const ui32 statId = blockIdx.z;

        __shared__ volatile double localBuffer[BlockSize];
        source += statId * statLineSize;


        const int minDocsPerBlock = BlockSize * 16;
        const int effectiveBlockCount = min(gridDim.x, (part.Size + minDocsPerBlock - 1) / minDocsPerBlock);

        double result = 0;

        if (blockIdx.x < effectiveBlockCount) {
            const int blockId = blockIdx.x % effectiveBlockCount;

            localBuffer[threadIdx.x] = ComputeSum <BlockSize>(source, part.Offset, part.Size, blockId, effectiveBlockCount);
            __syncthreads();

            result = FastInBlockReduce(threadIdx.x, localBuffer, BlockSize);
        }

        if (threadIdx.x == 0) {
            tempVars[gridDim.z * gridDim.y * blockIdx.x + blockIdx.y * gridDim.z + blockIdx.z] = result;
        }
    }


    void UpdatePartitionsProps(const TDataPartition* parts,
                               const ui32* partIds,
                               ui32 partCount,
                               const float* source,
                               ui32 statCount,
                               ui64 statLineSize,
                               ui32 tempVarsCount,
                               double* tempVars,
                               double* statSums,
                               TCudaStream stream
    ) {

        const ui32 blockSize = 512;

        dim3 numBlocks;

        numBlocks.y = partCount;
        numBlocks.z = statCount;
        numBlocks.x = CeilDivide(2 * TArchProps::SMCount(), (int)statCount);
        Y_VERIFY(numBlocks.x * numBlocks.y * numBlocks.z <= tempVarsCount);
        if (IsGridEmpty(numBlocks)) {
            return;
        }

        UpdatePartitionsPropsImpl<blockSize><<<numBlocks, blockSize, 0, stream>>>(partIds, parts, source, statLineSize, tempVars);
        {
            const ui32 saveBlockSize = 256;
            const ui32 numSaveBlocks = (numBlocks.y * numBlocks.z + saveBlockSize - 1) / saveBlockSize;
            SaveResultsImpl<<<numSaveBlocks, saveBlockSize, 0, stream>>>(partIds, tempVars, partCount, statCount, numBlocks.x, statSums);
        }
    }


    template <int BlockSize>
    __launch_bounds__(BlockSize, 2)
    __global__ void UpdatePartitionsPropsForSplitImpl(const ui32* leftPartIds,
                                                      const ui32* rightPartIds,
                                                      const TDataPartition* parts,
                                                      const float* source,
                                                      ui64 statLineSize,
                                                      double* tempVars) {
        const ui32 sourcePartCount = gridDim.y  / 2;
        const bool isLeft = blockIdx.y < sourcePartCount;
        const ui32* partIds = isLeft ? leftPartIds : rightPartIds;
        const ui32 leafId = partIds[isLeft ? blockIdx.y : blockIdx.y - sourcePartCount];
        TDataPartition part = parts[leafId];

        const ui32 statId = blockIdx.z;

        __shared__ volatile double localBuffer[BlockSize];
        source += statId * statLineSize;


        const int minDocsPerBlock = BlockSize;
        const int effectiveBlockCount = min(gridDim.x, (part.Size + minDocsPerBlock - 1) / minDocsPerBlock);

        double result = 0;

        if (blockIdx.x < effectiveBlockCount) {
            const int blockId = blockIdx.x % effectiveBlockCount;
            localBuffer[threadIdx.x] = ComputeSum<BlockSize, 4>(source, part.Offset, part.Size, blockId, effectiveBlockCount);
            __syncthreads();
            result = FastInBlockReduce(threadIdx.x, localBuffer, BlockSize);
        }

        if (threadIdx.x == 0) {
            tempVars[gridDim.z * gridDim.y * blockIdx.x + blockIdx.y * gridDim.z + blockIdx.z] = result;
        }
    }


    template <int BlockSize>
    __launch_bounds__(BlockSize, 2)
    __global__ void UpdatePartitionsPropsForSingleSplitImpl(const ui32 leftPartId,
                                                      const ui32 rightPartId,
                                                      const TDataPartition* parts,
                                                      const float* source,
                                                      ui64 statLineSize,
                                                      double* tempVars) {
//        const ui32 sourcePartCount = 2;
        const bool isLeft = blockIdx.y == 0;
        const ui32 leafId = isLeft ? leftPartId : rightPartId;
        TDataPartition part = parts[leafId];

        const ui32 statId = blockIdx.z;

        __shared__ volatile double localBuffer[BlockSize];
        source += statId * statLineSize;


        const int minDocsPerBlock = BlockSize;
        const int effectiveBlockCount = min(gridDim.x, (part.Size + minDocsPerBlock - 1) / minDocsPerBlock);

        double result = 0;

        if (blockIdx.x < effectiveBlockCount) {
            const int blockId = blockIdx.x % effectiveBlockCount;
            localBuffer[threadIdx.x] = ComputeSum<BlockSize, 4>(source, part.Offset, part.Size, blockId, effectiveBlockCount);
            __syncthreads();
            result = FastInBlockReduce(threadIdx.x, localBuffer, BlockSize);
        }

        if (threadIdx.x == 0) {
            tempVars[gridDim.z * gridDim.y * blockIdx.x + blockIdx.y * gridDim.z + blockIdx.z] = result;
        }
    }


    template <class TOutput>
    __global__ void SaveResultsForSplitImpl(const ui32* leftPartIds,
                                            const ui32* rightPartIds,
                                            const double* tempVars,
                                            ui32 partCount,
                                            ui32 statCount,
                                            int tempVarsBlockCount,
                                            TOutput* statSums) {

        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        const ui32 statId = i % statCount;
        const ui32 y =  i / statCount;
        if (i < partCount * statCount) {
            const ui32 leafId = (y < partCount / 2) ? leftPartIds[y] : rightPartIds[y - partCount / 2];
            double total = 0;
            for (int x = 0; x < tempVarsBlockCount; ++x) {
                total += __ldg(tempVars + i);
                tempVars += statCount * partCount;
            }
            statSums[leafId * statCount + statId] = total;
        }
    }

    template <class TOutput>
    __global__ void SaveResultsForSingleSplitImpl(const ui32 leftPartId,
                                                  const ui32 rightPartId,
                                                  const double* tempVars,
                                                  ui32 statCount,
                                                  int tempVarsBlockCount,
                                                  TOutput* statSums) {

        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        const ui32 statId = i % statCount;
        const ui32 y =  i / statCount;
        if (i < 2 * statCount) {
            const ui32 leafId = y == 0 ? leftPartId : rightPartId;
            double total = 0;
            for (int x = 0; x < tempVarsBlockCount; ++x) {
                total += __ldg(tempVars + i);
                tempVars += statCount * 2;
            }
            statSums[leafId * statCount + statId] = total;
        }
    }

    void UpdatePartitionsPropsForSplit(const TDataPartition* parts,
                                       const ui32* leftPartIds,
                                       const ui32* rightPartIds,
                                       ui32 partCount,
                                       const float* source,
                                       ui32 statCount,
                                       ui64 statLineSize,
                                       ui32 tempVarsCount,
                                       double* tempVars,
                                       double* statSums,
                                       TCudaStream stream) {

        const ui32 blockSize = 512;

        dim3 numBlocks;

        numBlocks.y = 2 * partCount;
        numBlocks.z = statCount;
        numBlocks.x = CeilDivide(2 * TArchProps::SMCount(), (int)statCount);
        Y_VERIFY(numBlocks.x * numBlocks.y * numBlocks.z <= tempVarsCount);
        if (IsGridEmpty(numBlocks)) {
            return;
        }

        UpdatePartitionsPropsForSplitImpl<blockSize><<<numBlocks, blockSize, 0, stream>>>(leftPartIds, rightPartIds, parts, source, statLineSize, tempVars);
        {
            const ui32 saveBlockSize = 256;
            const ui32 numSaveBlocks = (numBlocks.y * numBlocks.z + saveBlockSize - 1) / saveBlockSize;
            SaveResultsForSplitImpl<<<numSaveBlocks, saveBlockSize, 0, stream>>>(leftPartIds, rightPartIds, tempVars, 2 * partCount, statCount, numBlocks.x, statSums);
        }
    }

    void UpdatePartitionsPropsForSingleSplit(const TDataPartition* parts,
                                             const ui32 leftPartId,
                                             const ui32 rightPartId,
                                             const float* source,
                                             ui32 statCount,
                                             ui64 statLineSize,
                                             ui32 tempVarsCount,
                                             double* tempVars,
                                             double* statSums,
                                             TCudaStream stream) {
        const ui32 blockSize = 512;

        dim3 numBlocks;

        numBlocks.y = 2;
        numBlocks.z = statCount;
        numBlocks.x = CeilDivide(2 * TArchProps::SMCount(), (int)statCount);
        Y_VERIFY(numBlocks.x * numBlocks.y * numBlocks.z <= tempVarsCount);
        if (IsGridEmpty(numBlocks)) {
            return;
        }

        UpdatePartitionsPropsForSingleSplitImpl<blockSize><<<numBlocks, blockSize, 0, stream>>>(leftPartId, rightPartId, parts, source, statLineSize, tempVars);
        {
            const ui32 saveBlockSize = 256;
            const ui32 numSaveBlocks = (numBlocks.y * numBlocks.z + saveBlockSize - 1) / saveBlockSize;
            SaveResultsForSingleSplitImpl<<<numSaveBlocks, saveBlockSize, 0, stream>>>(leftPartId, rightPartId, tempVars, statCount, numBlocks.x, statSums);
        }

    }



    void UpdatePartitionsPropsForOffsets(const ui32* offsets,
                                         ui32 count,
                                         const float* source,
                                         ui32 statCount,
                                         ui64 statLineSize,
                                         ui32 tempVarsCount,
                                         double* tempVars,
                                         double* statSums,
                                         TCudaStream stream
    ) {
        const ui32 blockSize = 512;

        dim3 numBlocks;

        numBlocks.y = min(count, 65535);
        numBlocks.z = statCount;
        numBlocks.x = CeilDivide(2 * TArchProps::SMCount(), (int)statCount);
        Y_VERIFY((ui64)numBlocks.x * numBlocks.y * numBlocks.z <= tempVarsCount);
        if (IsGridEmpty(numBlocks)) {
            return;
        }

        UpdatePartitionsPropsForOffsetsImpl<blockSize><<<numBlocks, blockSize, 0, stream>>>(offsets, source,  statLineSize, count, tempVars);
        {
            const ui32 saveBlockSize = 256;
            const ui32 numSaveBlocks = (count * statCount + saveBlockSize - 1) / saveBlockSize;
            SaveResultsImpl<<<numSaveBlocks, saveBlockSize, 0, stream>>>(nullptr, tempVars, count, statCount, numBlocks.x, statSums);
        }
    }


    __global__ void FloatToDoubleImpl(const float* src, ui32 size, double* dst) {

        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            WriteThrough(dst + i, (double)__ldg(src + i));
        }
    }

    void CopyFloatToDouble(const float* src, ui32 size, double* dst, TCudaStream stream) {

        const ui32 blockSize = 128;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            FloatToDoubleImpl<<<numBlocks, blockSize, 0, stream>>>(src, size, dst);
        }
    }

    ui32 GetTempVarsCount(ui32 statCount, ui32 count) {
        return CeilDivide(2 * TArchProps::SMCount(), (int)statCount) * statCount * count;
    }
}
