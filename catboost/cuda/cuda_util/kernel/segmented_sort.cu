#include "segmented_sort.cuh"

#include <contrib/libs/nvidia/cub/cub/device/device_segmented_radix_sort.cuh>

namespace NKernel {

    template <typename K, typename V>
    hipError_t SegmentedRadixSort(K* keys, V* values,
                                   K* tmpKeys, V* tmpValues,
                                   int size,
                                   const ui32* segmentStarts, const ui32* segmentEnds,
                                   int numSegments,
                                   TSegmentedRadixSortContext& context, TCudaStream stream)
    {
        hipcub::DoubleBuffer<K> doubleBufferKeys(keys, tmpKeys);
        hipError_t error;
        int* starts = const_cast<int*>((const int*)(segmentStarts));
        int* ends = const_cast<int*>((const int*)(segmentEnds));

        if (values) {
            hipcub::DoubleBuffer<V> doubleBufferValues(values, tmpValues);

            if (context.Descending) {


                hipcub::DoubleBuffer<K> inputValues;
                error = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(context.TempStorage, context.TempStorageSize,
                                                                           doubleBufferKeys,
                                                                           doubleBufferValues,
                                                                           size,
                                                                           numSegments,
                                                                           starts, ends,
                                                                           context.FirstBit, context.LastBit,
                                                                           stream);
            } else {
                error = hipcub::DeviceSegmentedRadixSort::SortPairs(context.TempStorage, context.TempStorageSize,
                                                                 doubleBufferKeys,
                                                                 doubleBufferValues,
                                                                 size,
                                                                 numSegments,
                                                                 starts, ends,
                                                                 context.FirstBit, context.LastBit,
                                                                 stream);
            }

            if (doubleBufferValues.Current() != values) {
                hipMemcpyAsync(values, doubleBufferValues.Current(), sizeof(V) * size, hipMemcpyDefault, stream);
            }
        } else {
            if (context.Descending) {
                error = hipcub::DeviceSegmentedRadixSort::SortKeysDescending(context.TempStorage, context.TempStorageSize,
                                                                          doubleBufferKeys,
                                                                          size,
                                                                          numSegments,
                                                                          starts, ends,
                                                                          context.FirstBit, context.LastBit,
                                                                          stream);
                } else {
                    error = hipcub::DeviceSegmentedRadixSort::SortKeys(context.TempStorage, context.TempStorageSize,
                                                                    doubleBufferKeys,
                                                                    size,
                                                                    numSegments,
                                                                    starts, ends,
                                                                    context.FirstBit, context.LastBit,
                                                                    stream);
                }
        }
        //TODO(noxoomo): error handling
        if (doubleBufferKeys.Current() != keys) {
            hipMemcpyAsync(keys, doubleBufferKeys.Current(), sizeof(K) * size, hipMemcpyDefault, stream);
        }
        return error;
    }


    #define SEGMENTED_RADIX_SORT(Type) \
    template hipError_t SegmentedRadixSort(Type* keys, Type* values, Type* tmpKeys, Type* tmpValues, int size, \
                                            const ui32* segmentStarts, const ui32* segmentEnds, int segmentsCount, \
                                            TSegmentedRadixSortContext& context, TCudaStream stream);

    SEGMENTED_RADIX_SORT(float)
    SEGMENTED_RADIX_SORT(ui32)

}
