#include <catboost/cuda/cuda_util/kernel/sort_templ.cuh>

namespace NKernel {
    template hipError_t RadixSort(ui32* keys, ui32* values, ui32 size, TRadixSortContext& context,  TCudaStream stream);
    template hipError_t RadixSort(ui32* keys, ui64* values, ui32 size, TRadixSortContext& context,  TCudaStream stream);
}
