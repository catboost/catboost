#include "hip/hip_runtime.h"
#include "random.cuh"
#include "random_gen.cuh"
#include <library/cpp/cuda/wrappers/arch.cuh>

namespace NKernel {

    __global__ void PoissonRandImpl(ui64* seeds, ui32 seedSize,
                                    const float* alpha, int* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextPoisson(&s, alpha[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void PoissonRand(ui64* seeds, ui32 size, const float* alphas, int* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        PoissonRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, alphas, result);
    }

    __global__ void GaussianRandImpl(ui64* seeds, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextNormal(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GaussianRand(ui64* seeds, ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GaussianRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void UniformRandImpl(ui64* seeds, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextUniform(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void UniformRand(ui64* seeds, ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        UniformRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void GammaRandImpl(ui64* seeds, const float* alphas,
                                  const float* scale, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = __ldg(seeds + i);
            result[i] = NextGamma(&s, alphas[i], scale[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GammaRand(ui64* seeds, const float* alphas, const float* scale,
                   ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GammaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, scale, size, result);
    }

    __global__ void BetaRandImpl(ui64* seeds, const float* alphas,
                                 const float* betas, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextBeta(&s, alphas[i], betas[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void BetaRand(ui64* seeds, const float* alphas, const float* betas,
                  ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        BetaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, betas, size, result);
    }


    __global__ void GenerateSeedsImpl(ui64 baseSeed, ui64* seeds, ui64 size) {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size) {
            ui32 baseSeed1 = (baseSeed >> 32);
            ui32 baseSeed2 = (baseSeed & 0xFFFFFF);
            ui32 tmp1 = 134775813 * i + 1664525 * baseSeed1 + 69069 *  baseSeed2 + 1013904225;
            ui32 tmp2 = 1664525 * (baseSeed1 + 134775813  * baseSeed2 + i + 1) + 1013904223;
            for (int j = 0; j < 4 + (threadIdx.x % 8); ++j) {
                tmp1 = AdvanceSeed32(&tmp1);
                tmp2 = AdvanceSeed32(&tmp2);
            }
            //no math here, just stupid heuristics
            ui64 s = (((ui64)tmp1) << 32) | tmp2;
            seeds[i] = AdvanceSeed(&s, blockIdx.x);
            i += gridDim.x * blockDim.x;
        }
    }

    void GenerateSeeds(ui64 baseSeed, ui64* seeds, ui64 size, TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   (ui64)TArchProps::MaxBlockCount());
        GenerateSeedsImpl<<<numBlocks, blockSize, 0, stream>>>(baseSeed, seeds, size);

    }


}
