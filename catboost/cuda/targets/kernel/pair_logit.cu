#include "hip/hip_runtime.h"
#include "pair_logit.cuh"
#include <catboost/cuda/cuda_lib/kernel/kernel.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <catboost/cuda/cuda_util/kernel/fill.cuh>


namespace NKernel {


    template <int BLOCK_SIZE>
    __global__ void PairLogitPointwiseTargetImpl(const float* point,
                                                 const uint2* pairs, const float* pairWeights,
                                                 const ui32* writeMap,
                                                 ui32 pairCount, int pairShift,
                                                 float* functionValue,
                                                 float* der,
                                                 float* der2)  {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ float tmpScores[BLOCK_SIZE];

        uint2 pair;
        if (i < pairCount) {
            pair = __ldg(pairs + i);
        } else {
            pair.x = pairShift;
            pair.y = pairShift;
        }
        pair.x -= pairShift;
        pair.y -= pairShift;
        const float w = pairWeights && (i < pairCount) ? pairWeights[i] : 1.0f;
        const float diff = i < pairCount ? __ldg(point + pair.x) - __ldg(point + pair.y) : 0;
        const float expDiff = __expf(diff);
        const float p = max(min(isfinite(1.0f + expDiff) ? expDiff / (1.0f + expDiff) : 1.0f, 1.0f - 1e-40f), 1e-40f);

        const float direction = (1.0f - p);

        const ui32 firstDst = writeMap ? writeMap[pair.x] : pair.x;
        const ui32 secondDst = writeMap ? writeMap[pair.y] : pair.y;

        if (der && i < pairCount) {
            atomicAdd(der + firstDst, w * direction);
            atomicAdd(der + secondDst, -w * direction);
        }

        if (der2 && i < pairCount) {
            const float scale = p * (1.0f - p);
            atomicAdd(der2 + firstDst, w * scale);
            atomicAdd(der2 + secondDst, w * scale);
        }

        if (functionValue) {
            const float logExpValPlusOne = isfinite(1.0f + expDiff) ? __logf(1.0f + expDiff) : diff;
            tmpScores[threadIdx.x] = (i < pairCount) ? w * (diff - logExpValPlusOne) : 0;

            __syncthreads();
            float val = FastInBlockReduce<float>(threadIdx.x, tmpScores, BLOCK_SIZE);
            if (threadIdx.x == 0) {
                atomicAdd(functionValue, val);
            }
        }
    }


    __global__ void MakePairWeightsImpl(const uint2* pairs, const float* pairWeights, ui32 pairCount,
                                        float* weights)  {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < pairCount) {
            uint2 pair = __ldg(pairs + i);
            const float w = pairWeights ? pairWeights[i] : 1.0f;
            atomicAdd(weights + pair.x, w);
            atomicAdd(weights + pair.y, w);
        }
    }

    void MakePairWeights(const uint2* pairs, const float* pairWeights, ui32 pairCount,
                         float* weights, TCudaStream stream) {
        const int blockSize = 512;
        const int numBlocks = (pairCount + blockSize - 1) / blockSize;
        MakePairWeightsImpl<<<numBlocks, blockSize, 0, stream>>>(pairs, pairWeights, pairCount, weights);

    }

    void PairLogitPointwiseTarget(const float* point,
                                  const uint2* pairs, const float* pairWeights,
                                  const ui32* writeMap,
                                  ui32 pairCount, int pairShift,
                                  float* functionValue,
                                  float* der,
                                  float* der2,
                                  ui32 docCount,
                                  TCudaStream stream) {

        const int blockSize = 1024;
        const int numBlocks = (pairCount + blockSize - 1) / blockSize;
        if (functionValue) {
            FillBuffer(functionValue, 0.0f, 1, stream);
        }
        if (der) {
            FillBuffer(der, 0.0f, docCount, stream);
        }
        if (der2) {
            FillBuffer(der2, 0.0f, docCount, stream);
        }
        if (numBlocks)
        {
            PairLogitPointwiseTargetImpl<blockSize> << <numBlocks, blockSize, 0, stream >> > (point, pairs, pairWeights, writeMap, pairCount, pairShift, functionValue, der, der2);
        }

    }



    template <int BLOCK_SIZE>
    __global__ void PairLogitPairwiseImpl(const float* point,
                                          const uint2* pairs,
                                          const float* pairWeights,
                                          ui32 pairCount,
                                          const ui32* scatterDerIndices,
                                          float* functionValue,
                                          float* pointDer,
                                          float* pairsDer2)  {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        uint2 pair;
        if (i < pairCount) {
            pair = __ldg(pairs + i);
        }
        const float w = pairWeights && (i < pairCount) ? pairWeights[i] : 1.0f;
        const float diff = i < pairCount ? __ldg(point + pair.x) - __ldg(point + pair.y) : 0;
        const float expDiff = __expf(diff);
        const float p = ClipProb(isfinite(expDiff + 1.0f) ? expDiff / (1.0f + expDiff) : 1.0f);
        const float direction = w * (1.0f - p);
        const float pairDer2 = w * p * (1.0f - p);

        if (i < pairCount) {
            const ui32 pairx = scatterDerIndices == nullptr ? pair.x : scatterDerIndices[pair.x];
            const ui32 pairy = scatterDerIndices == nullptr ? pair.y : scatterDerIndices[pair.y];
            atomicAdd(pointDer + pairx, direction);
            atomicAdd(pointDer + pairy, -direction);
            if (pairsDer2) {
                pairsDer2[i] = pairDer2;
            }
        }

        if (functionValue) {
            const float logExpValPlusOne = isfinite(expDiff + 1.0f) ? __logf(1.0f + expDiff) : diff;
            __shared__  float scores[BLOCK_SIZE];
            scores[threadIdx.x] = (i < pairCount) ? w * (diff - logExpValPlusOne) : 0;
            __syncthreads();

            float val = FastInBlockReduce<float>(threadIdx.x, scores, BLOCK_SIZE);
            if (threadIdx.x == 0) {
                atomicAdd(functionValue, val);
            }
        }
    }


    void PairLogitPairwise(const float* point,
                           const uint2* pairs,
                           const float* pairWeights,
                           const ui32* scatterDerIndices,
                           float* value,
                           float* pointDer,
                           ui32 docCount,
                           float* pairDer2,
                           ui32 pairCount,
                           TCudaStream stream) {

        const int blockSize = 256;
        const int numBlocks = (pairCount + blockSize - 1) / blockSize;
        FillBuffer(pointDer, 0.0f, docCount, stream);
        if (value != nullptr) {
            FillBuffer(value, 1.0f, 1, stream);
        }
        if (numBlocks) {
            PairLogitPairwiseImpl<blockSize> << <numBlocks, blockSize, 0, stream >> > (point, pairs, pairWeights, pairCount, scatterDerIndices,  value, pointDer, pairDer2);
        }
    }

    __global__ void RemoveOffsetsBiasImpl(ui32 bias,
                                          ui32 nzPairCount,
                                          uint2* nzPairs) {

        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        while (i < nzPairCount) {
            uint2 pair = nzPairs[i];
            pair.x -= bias;
            pair.y -= bias;
            nzPairs[i] = pair;
            i += blockDim.x * gridDim.x;
        }
    }

    void RemoveOffsetsBias(ui32 bias,
                           ui32 nzPairCount,
                           uint2* nzPairs,
                           TCudaStream stream) {

        const int blockSize = 256;
        const int numBlocks = (nzPairCount + blockSize - 1) / blockSize;
        if (numBlocks > 0) {
            RemoveOffsetsBiasImpl<<< numBlocks, blockSize, 0, stream >>> (bias, nzPairCount, nzPairs);
        }
    }
}
