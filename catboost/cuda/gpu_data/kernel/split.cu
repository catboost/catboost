#include "hip/hip_runtime.h"
#include <catboost/cuda/gpu_data/gpu_structures.h>

#include <catboost/cuda/cuda_util/kernel/compression.cuh>
#include <catboost/cuda/cuda_util/kernel/compression_helper.cuh>
#include <library/cpp/cuda/wrappers/arch.cuh>

namespace NKernel {

    struct TBinSplitLoader {
        const ui32* CompressedIndex;
        const ui32* Indices;
        ui32 Value;
        ui32 Mask;
        bool TakeEqual;

        __forceinline__ __device__ TBinSplitLoader(const ui32* index,
                                                   const ui32* indices,
                                                   const ui32 value,
                                                   const ui32 mask,
                                                   bool takeEqual)
                : CompressedIndex(index)
                , Indices(indices)
                , Value(value)
                , Mask(mask)
                , TakeEqual(takeEqual) {

        }

        __forceinline__ __device__ ui32 operator()(ui32 offset) {
            const ui32 idx = Indices ? Indices[offset] : offset;
            const ui32 featureVal = CompressedIndex[idx] & Mask;
            return static_cast<ui32>(TakeEqual ? (featureVal == Value) : featureVal > Value);
        }
    };


    struct TFloatSplitLoader {

        const float* Values;
        const ui32* Indices;
        float Border;

        __device__ TFloatSplitLoader(const float* values,
                                     const ui32* indices,
                                     float border
        )
                : Values(values)
                , Indices(indices)
                , Border(border) {

        }

        __forceinline__ __device__ ui32 operator()(ui32 offset) {
            ui32 idx = Indices ? Indices[offset] : offset;
            return static_cast<ui32>(Values[idx] > Border);
        }
    };


    struct TBinUpdater {
        ui32* Bins;
        ui32 Depth;

        __forceinline__ __device__ TBinUpdater(ui32* bins, ui32 depth)
                : Bins(bins)
                , Depth(depth) {

        }

        __forceinline__ __device__ ui32 operator()(ui32 offset, ui32 bin) {
            return Bins[offset] |= bin << Depth;
        }
    };


    template <int BLOCK_SIZE>
    __global__ void WriteCompressedSplitImpl(TCFeature feature, ui32 binIdx,
                                             const ui32* compressedIndex,
                                             const ui32* indices, int size,
                                             ui64* compressedBits)
    {
        TCompressionHelper<ui64, BLOCK_SIZE> helper(1);

        if (indices) {
            indices += helper.KeysPerBlock() * blockIdx.x;
        } else {
            compressedIndex +=  helper.KeysPerBlock() * blockIdx.x;
        }
        size -= helper.KeysPerBlock() * blockIdx.x;

        compressedBits += BLOCK_SIZE * blockIdx.x;
        compressedIndex += feature.Offset;

        const ui32 value = binIdx << feature.Shift;
        const ui32 mask = feature.Mask << feature.Shift;

        TBinSplitLoader loader(compressedIndex, indices, value, mask, feature.OneHotFeature);
        helper.CompressBlock(loader, size, compressedBits);
    }



    template <int BLOCK_SIZE>
    __global__ void WriteCompressedSplitFloatImpl(const float* values, float border,
                                                  const ui32* indices, int size,
                                                  ui64* compressedBits)
    {
        TCompressionHelper<ui64, BLOCK_SIZE> helper(1);

        if (indices) {
            indices += helper.KeysPerBlock() * blockIdx.x;
        } else {
            values += helper.KeysPerBlock() * blockIdx.x;
        }
        size -= helper.KeysPerBlock() * blockIdx.x;
        compressedBits += BLOCK_SIZE * blockIdx.x;

        TFloatSplitLoader loader(values, indices, border);
        helper.CompressBlock(loader, size, compressedBits);
    }


    template <int BLOCK_SIZE>
    __global__ void UpdateBinsImpl(const ui64* compressedBits,
                                   ui32 depth,
                                   ui32* bins, int size) {

        TCompressionHelper<ui64, BLOCK_SIZE> helper(1);

        bins += helper.KeysPerBlock() * blockIdx.x;
        size -= helper.KeysPerBlock() * blockIdx.x;
        compressedBits += BLOCK_SIZE * blockIdx.x;

        TBinUpdater writer(bins, depth);
        helper.DecompressBlock(writer, compressedBits, size);
    }

    void WriteCompressedSplit(TCFeature feature, ui32 binIdx,
                              const ui32* compressedIndex,
                              const ui32* indices, int size,
                              ui64* compressedBits,
                              TCudaStream stream) {

        constexpr int blockSize = CompressCudaBlockSize();
        const int numBlocks = CeilDivide(size, TCompressionHelper<ui64, blockSize>(1).KeysPerBlock());

        if (numBlocks) {
            WriteCompressedSplitImpl<blockSize> << < numBlocks, blockSize, 0, stream >> >(feature, binIdx, compressedIndex,
                    indices, size, compressedBits);
        }
    }

    void WriteCompressedSplitFloat(const float* values, float border,
                                   const ui32* indices, int size,
                                   ui64* compressedBits,
                                   TCudaStream stream) {
        constexpr int blockSize = CompressCudaBlockSize();
        const int numBlocks = CeilDivide(size, TCompressionHelper<ui64, blockSize>(1).KeysPerBlock());

        if (numBlocks) {
            WriteCompressedSplitFloatImpl<blockSize> << < numBlocks, blockSize, 0, stream >> >(values, border, indices, size, compressedBits);
        }
    }

    void UpdateBins(const ui64* compressedBits,
                    ui32 depth,
                    ui32* bins, int size,
                    TCudaStream stream) {

        constexpr int blockSize = CompressCudaBlockSize();
        const int numBlocks = CeilDivide(size, TCompressionHelper<ui64, blockSize>(1).KeysPerBlock());

        if (numBlocks) {
            UpdateBinsImpl<blockSize> << < numBlocks, blockSize, 0, stream >> >(compressedBits, depth, bins, size);
        }
    }


    __global__ void UpdateBinsFromCompressedIndexImpl(const ui32* compressedIndex,
                                                      const ui32* indices,
                                                      const int size,
                                                      const TCFeature feature,
                                                      const ui32 binIdx,
                                                      const ui32 depth,
                                                      ui32* bins)
    {

        compressedIndex += feature.Offset;
        int i =  blockIdx.x * blockDim.x + threadIdx.x;

        const ui32 value = binIdx << feature.Shift;
        const ui32 mask = feature.Mask << feature.Shift;

        while (i < size) {
            const ui32 idx = indices ? __ldg(indices + i) : i;
            const ui32 featureVal = __ldg(compressedIndex + idx) & mask;
            const ui32 split = (feature.OneHotFeature ? (featureVal == value) : featureVal > value);
            bins[i] |= split << depth;
            i += blockDim.x * gridDim.x;
        }
    }

    void UpdateBinsFromCompressedIndex(const ui32* compressedIndex,
                                       const ui32* indices,
                                       const int size,
                                       const TCFeature feature,
                                       const ui32 binIdx,
                                       const ui32 depth,
                                       ui32* bins,
                                       TCudaStream stream) {

        constexpr int blockSize = 256;
        const int numBlocks = min(CeilDivide(size, blockSize), TArchProps::MaxBlockCount());

        if (numBlocks) {
            UpdateBinsFromCompressedIndexImpl << < numBlocks, blockSize, 0, stream >> >(compressedIndex, indices, size, feature, binIdx, depth, bins);
        }
    }

}
